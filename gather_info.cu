#include "hip/hip_runtime.h"
#include <iostream>

__global__ void add( int a, int b, int *c ) {
*c = a + b;
}
int main( void ) {
int c;
int *dev_c;
hipMalloc( (void**)&dev_c, sizeof(int) ) ;
add<<<1,1>>>( 2, 7, dev_c );
hipMemcpy( &c,dev_c,sizeof(int),hipMemcpyDeviceToHost ) ;
printf( "2 + 7 = %d\n", c ); //example
int count;
printf("%d\n",hipGetDeviceCount( &count ));
hipFree( dev_c );
//Collect device information
hipDeviceProp_t prop;
hipGetDeviceProperties( &prop, 0) ;
printf("%d\n",prop);
printf( " --- General Information for device %d ---\n", 0 );
printf( "Name: %s\n", prop.name );
printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
printf( "Clock rate: %d\n", prop.clockRate );
printf( "Device copy overlap: " );
if (prop.deviceOverlap)
printf( "Enabled\n" );
else
printf( "Disabled\n" );
printf( "Kernel execition timeout : " );
if (prop.kernelExecTimeoutEnabled)
printf( "Enabled\n" );
else
printf( "Disabled\n" );
printf( " --- Memory Information for device %d ---\n", 0);
printf( "Total global mem: %ld\n", prop.totalGlobalMem );
printf( "Total constant Mem: %ld\n", prop.totalConstMem );
printf( "Max mem pitch: %ld\n", prop.memPitch );
printf( "Texture Alignment: %ld\n", prop.textureAlignment );
printf( " --- MP Information for device %d ---\n", 0 );
printf( "Multiprocessor count: %d\n",
prop.multiProcessorCount );
printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
printf( "Registers per mp: %d\n", prop.regsPerBlock );
printf( "Threads in warp: %d\n", prop.warpSize );
printf( "Max threads per block: %d\n",
prop.maxThreadsPerBlock );
printf( "Max thread dimensions: (%d, %d, %d)\n",
prop.maxThreadsDim[0], prop.maxThreadsDim[1],
prop.maxThreadsDim[2] );
printf( "Max grid dimensions: (%d, %d, %d)\n",
prop.maxGridSize[0], prop.maxGridSize[1],
prop.maxGridSize[2] );
printf( "\n" );

return 0;
}
