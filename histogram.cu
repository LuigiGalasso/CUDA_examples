
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <algorithm>
#include <math.h>
#include <fstream>

using namespace std;

//number of bins of the plot
const int BINS = 7;
const int DIV  = ((26+BINS -1)/BINS); //alphabet is composed of 26 letters

//GPU kernel for coputing a histogram
//a:array in global memory
//N:size of array

__global__ void histogram(char *a, int *result, int N){
	int tid = threadIdx.x+ blockIdx.x*blockDim.x;
	//bin position where threads are grouped together
	int alpha_position;
	for(int i = tid;i<N;i+=(blockDim.x * gridDim.x)){
	//position in the alphabet subtracting ASCII value
		alpha_position = a[i] - 'a';
		atomicAdd(&result[alpha_position/DIV],1);
	}
}

int main(){
	int N = 1 << 22;
	char vector [N];
	int result [BINS];
	//initialize array
	for (int i = 0;i < N;i++) vector[i] = 'a' + rand()%26;

	
	char *input;
	int *output;
	hipMalloc( (void**)&input, N*sizeof(char) ) ;
	hipMalloc( (void**)&output, BINS*sizeof(int) ) ;
	
	hipMemcpy( input,vector,N*sizeof(char),hipMemcpyHostToDevice ) ;
	
	int THREADS = 512;
	int BLOCKS = (N + THREADS -1)/THREADS;
	
	histogram<<<BLOCKS,THREADS>>>(input,output,N); 
	hipMemcpy(result,output,BINS*sizeof(int),hipMemcpyDeviceToHost ) ;
	for(int i=0;i<BINS;i++) printf("BINS %d : %d\n",i,result[i]);
	hipFree(input);
	hipFree(output);


	return 0;


}
