
#include <hip/hip_runtime.h>
#include <iostream>

//grid composed of blocks composed of threads
__global__ void add(int *a, int *b, int *c, int v) {
	int tid = threadIdx.x+ blockIdx.x*blockDim.x;
	while(tid < v){
		c[tid]= a[tid] +b[tid];
		tid += blockDim.x*gridDim.x;//after thread finishes tid is incremented of the the total number of threads running in the grid to start working on the next element
	}
}

int main( void ) {
	int v = 123476;// not power of 2 need to pad the dimensions 
	int m = 1<<15;
	int a[v],b[v], c[v];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc( (void**)&dev_a, v*sizeof(int) ) ;
	hipMalloc( (void**)&dev_b, v*sizeof(int) ) ;
	hipMalloc( (void**)&dev_c, v*sizeof(int) ) ;
	
	for(int i=0;i<v;i++) {
		a[i] = i;
		if(i < m ) b[i] = i^2;
		else b[i]=0;	
	}
	hipMemcpy( dev_a,a,v*sizeof(int),hipMemcpyHostToDevice ) ;
	hipMemcpy( dev_b,b,v*sizeof(int),hipMemcpyHostToDevice ) ;

	int nthreadPerBlock = 128;
	int nblock = (v+nthreadPerBlock-1)/nthreadPerBlock;
	add<<<nblock,nthreadPerBlock>>>( dev_a, dev_b, dev_c,v );

	hipMemcpy( &c,dev_c,v*sizeof(int),hipMemcpyDeviceToHost ) ;
	
	for(int i=0;i<v;i++){
		printf("%d + %d = %d\n",a[i],b[i],c[i]);	
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}
